#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <math.h>
#include <string>
#include <cutil_inline.h>
using namespace std;
typedef struct Matrix{
	float * ptr;
	int row;
	int col;
	char flag;
} M;



#define getitem(M1,i,j) (((j)==0 && ((M1).flag)==-1)? 1:(M1).ptr[(i)*(M1.col)+(j)+(M1).flag])
#define sigmoid(x) (1.0f/(1.0f+exp(-(x))))
__device__ float mul(M M1,M M2,int threadid){
	float a=0.0f;
	for(int i=0;i<(M1).col;i++){
		a+=getitem( M1,threadid/M1.col,i)*getitem(M2,i,threadid%M2.row);
	}
	return a;
}
__device__ float mul_(M M1,M M2,int threadid){
	float a=0.0f;
	for(int i=0;i<(M1).col;i++){
		a+=getitem( M1,threadid/M1.col,i)*getitem(M2,threadid%M2.row,i);
	}
	return a;
}
__device__ float _mul(M M1,M M2,int threadid){
	float a=0.0f;
	for(int i=0;i<(M1).row;i++){
		a+=getitem( M1,i,threadid/M1.col)*getitem(M2,i,threadid%M2.row);
	}
	return a;
}
//参数a2是虚拟化之后的，a3是正常的
__global__ void nnCostFunction(M theta1,M theta2,M X,int * Y,float lambda,M a2,M a3,int num) {
	int gid = blockIdx.x*blockDim.x+threadIdx.x;
	M a1 = X;a1.col--;a1.flag=-1;
for(int iter=0;iter<num;iter++){
	if(gid<(a2.col-1)*a2.row){
		a2.ptr[gid] = sigmoid(mul_(a1,theta1,gid));
	}
	__syncthreads();
	//以下计算可以再写一个函数
	if(gid<(a3.col)*a3.row){
		a3.ptr[gid] = sigmoid(mul_(a2,theta2,gid));
	}
	__syncthreads();//在这里，又进行了一次map
	if(gid<a3.row){
		a3.ptr[gid*a3.col+Y[gid]]-=1;
	}
	__syncthreads();
	if(gid<theta2.row*theta2.col){
		theta2.ptr[gid]-=(_mul(a3,a2,gid) + (gid%theta2.row) ? (lambda*theta2.ptr[gid]) : 0.0f)/(float)X.row;
	}
	__syncthreads();
	a2.col--;a2.flag=0;
	M theta2_1=theta2;theta2_1.col--;theta2_1.flag=1;
	if(gid<a2.row*a2.col){
		float temp = a2.ptr[gid];
		a2.ptr[gid]=(mul(a3,theta2_1,gid)*temp*(1-temp));
	}
	__syncthreads();
	if(gid<theta1.col*theta1.row){
		theta1.ptr[gid]-=(_mul(a2,a1,gid)+(gid%theta1.row)?(lambda*theta1.ptr[gid]):0.0f)/(float)X.row;
	}
	__syncthreads();
	a2.flag=-1;a2.col++;
}

}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */

void initdatafloat(char * filename, unsigned int size, float * p){
    fstream file(filename,ios::in);
    for (unsigned int i = 0; i<size; i++) {
        file >> p[i];
    }
    file.close();
}
void initdataint(char * filename, unsigned int size, int * p){
    fstream file(filename,ios::in);
    if(!file){return ;}
    for (unsigned int i = 0; i<size; i++) {
        file >> p[i];
    }
    file.close();
}

void inittheta(int len,float* p){
	for(int i=0;i<len;i++){
		p[i]=0.12f*(((float)(rand()%1000))/1000.0f*2-1);
	}
}
void printtofile(char * filename,int len,int *ptr ){
    fstream file(filename,ios::out);
    for (int i = 0; i<len; i++) {
        file << ptr[i] << " ";
        file << endl;
    }
    file.close();
}
int main(void) {
	printf("fuck");
	M  h_X, h_theta1, h_theta2,d_X, d_theta1,d_theta2,d_a3,d_a2;
	h_X.ptr=new float[5000*400];h_X.row=5000;h_X.col=400;h_X.flag=0;
	h_theta1.ptr=new float[401*25];h_theta1.row=401;h_theta1.col=25;h_theta1.flag=0;
	h_theta2.ptr= new float[26*10];h_theta2.row=26;h_theta2.col=10;h_theta2.flag=0;
	d_a3.ptr=0;d_a3.row=5000;d_a3.col=10;d_a3.flag=0;
	d_a2.ptr=0;d_a2.row=5000;d_a2.col=26;d_a2.flag=-1;
	int * h_yptr,* d_yptr;

	initdatafloat("Y.dat", 5000, h_yptr);
    initdatafloat("X2.dat", h_X.row*h_X.col, h_X.ptr);

    inittheta(401*25,h_theta1.ptr);
    inittheta(26*10,h_theta2.ptr);
    d_X=h_X;d_theta1=h_theta1;d_theta2=h_theta2;
    cudaSafeCall(hipMalloc((void**) &d_X.ptr, sizeof(float) *d_X.col*d_X.row ));
    cudaSafeCall(hipMalloc((void**) &d_yptr, sizeof(int) *5000 ));
    cudaSafeCall(hipMalloc((void**) &d_theta1.ptr, sizeof(float) *d_theta1.col*d_theta1.row ));
    cudaSafeCall(hipMalloc((void**) &d_theta2.ptr, sizeof(float) *d_theta2.col*d_theta2.row ));
    cudaSafeCall(hipMalloc((void**) &d_a2.ptr, sizeof(float) *5000*25 ));
    cudaSafeCall(hipMalloc((void**) &d_a3.ptr, sizeof(float) *d_a3.col*d_a3.row ));

    cudaSafeCall(hipMemcpy(d_X.ptr, h_X.ptr, sizeof(float)*d_X.col*d_X.row,hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(d_yptr, h_yptr, sizeof(float)*5000,hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(d_theta1.ptr, h_theta1.ptr, sizeof(float)*d_theta1.col*d_theta1.row,hipMemcpyHostToDevice));
    cudaSafeCall(hipMemcpy(d_theta2.ptr, h_theta2.ptr, sizeof(float)*d_theta2.col*d_theta2.row,hipMemcpyHostToDevice));
    cout<<"fuck";
    nnCostFunction<<<5000*25/1024+1,1024>>>(d_theta1,d_theta2,d_X,d_yptr,0.1f,d_a2,d_a3,1);


//	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
//	CUDA_CHECK_RETURN(hipGetLastError());
//	CUDA_CHECK_RETURN(hipMemcpy(odata, d, sizeof(int) * WORK_SIZE, hipMemcpyDeviceToHost));
//
//	for (i = 0; i < WORK_SIZE; i++)
//		printf("Input value: %u, device output: %u\n", idata[i], odata[i]);
//
//	CUDA_CHECK_RETURN(hipFree((void*) d));
//	CUDA_CHECK_RETURN(hipDeviceReset());
    cudaSafeCall(hipFree(d_X.ptr));
    cudaSafeCall(hipFree(d_theta1.ptr));
    cudaSafeCall(hipFree(d_theta2.ptr));
    cudaSafeCall(hipFree(d_a2.ptr));
    cudaSafeCall(hipFree(d_a3.ptr));
    cudaSafeCall(hipFree(d_yptr));
    delete [] h_X.ptr;
    delete [] h_theta1.ptr;
    delete [] h_theta2.ptr;
    delete [] h_yptr;
    cout<<"fuck";
	return 0;
}
